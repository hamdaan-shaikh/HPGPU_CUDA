#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <iostream>
#include <stdexcept>
#include <limits>

template<int TBlocksize, typename T>
__global__
void kernel_reduce(T* x, T* y, int n)
{
  
  // TODO:
  // with Global memory
  // - each thread starts with x[i] and sums it up with x[i+grid], ...
  // - atomicAdd the results into y

  // with Shared Memory and without atomics
  // - sum up the elements stored in x
  // and compute the partial results as done in the first task with global memory
  // - store it into shared memory
  // - perform a block and warp reduce (can be described as one loop)
  // - either use atomicAdd or a second pass, where block results are summed up

  // --------
  // Level 1: grid reduce
  // --------

  // --------
  // Level 2: block + warp reduce (on shared memory)
  // --------
  // you might want to assume TBlocksize to be power-of-2 to save some checks


  // TODO:
  // store block result to gmem (only one thread per block should do this)
}

template<typename T, int TRuns, int TBlocksize>
void reduce(T init, size_t n, int dev) {

  CHECK_CUDA( hipSetDevice(dev) );
  hipDeviceProp_t prop;
  CHECK_CUDA( hipGetDeviceProperties(&prop, dev) );
  hipEvent_t cstart, cend;
  CHECK_CUDA(hipEventCreate(&cstart));
  CHECK_CUDA(hipEventCreate(&cend));

  std::cout << getCUDADeviceInformations(dev).str()
            << "\n\n";

  // for grid-striding loops get number of SMs
  int numSMs;
  CHECK_CUDA( hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, dev) );
  dim3 blocks( 16*numSMs ); // 16*128 = 2048 (=max resident threads on SM), rule of thumb
  if( blocks.x > ((n-1)/TBlocksize+1) )
    blocks.x = (n-1)/TBlocksize+1;

  T* h_x = new T[n]; // [on host]
  T* x = nullptr; // input data [on device]
  T* y = nullptr; // result [on device]
  T result_gpu = 0; // final result [on host]

  // TODO: allocate memory
  // We allocate the host size of n for x, and unit size for y
  CHECK_CUDA ( hipMalloc(&x,n*sizeof(T)) ); 
  CHECK_CUDA ( hipMalloc(&y,sizeof(T)) );

  // init host memory
  for (int i = 0; i < n; i++) {
    h_x[i] = init;
  }

  // TODO: transfer data to GPU
  CHECK_CUDA ( hipMemcpy(x, h_x, n*sizeof(T), hipMemcpyHostToDevice) );

  // time measurement
  float milliseconds = 0;
  float min_ms = std::numeric_limits<float>::max();

  for(int r=0; r<TRuns; ++r) {
    CHECK_CUDA ( hipMemset( y, 0 , sizeof(T) ) );
    CHECK_CUDA(hipEventRecord(cstart));

    // TODO: call the kernel (maybe second pass needed for summing up partial results)
    //  kernel_reduce<TBlocksize><<<blocks, TBlocksize>>>( ...
    // Attention: do not write to x as we call this part several times for benchmark
    

    CHECK_CUDA( hipEventRecord(cend) );
    CHECK_CUDA( hipEventSynchronize(cend) );
    CHECK_CUDA( hipGetLastError() );
    CHECK_CUDA( hipEventElapsedTime(&milliseconds, cstart, cend) );
    if(milliseconds<min_ms)
      min_ms = milliseconds;
  }

  // TODO: get the result
  CHECK_CUDA ( hipMemcpy(y, x, sizeof(T), hipMemcpyHostToDevice) );
  CHECK_CUDA ( hipMemcpy(result_gpu, y, sizeof(T), hipMemcpyDeviceToHost) );

  // output
  std::cout << "Result (n = "<<n<<"):\n"
            << "GPU: " << result_gpu << " (min kernels time = "<< min_ms <<" ms)\n"
            << "expected: " << init*n <<"\n"
            << (init*n != result_gpu ? "MISMATCH!!" : "Success") << "\n"
            << "max bandwidth: "<<n*sizeof(T)/min_ms*1e-6<<" GB/s"
            << std::endl;

  delete[] h_x;
  CHECK_CUDA(hipFree(x));
  CHECK_CUDA(hipFree(y));
  CHECK_CUDA(hipEventDestroy(cstart));
  CHECK_CUDA(hipEventDestroy(cend));
}

int main(int argc, const char** argv)
{
  int dev=0;
  int n = 0;
  if(argc==2)
    n = atoi(argv[1]);
  if(n<2)
    n = 1<<28; // = 2^28
  try{
    // reduce<data type, number of benchmark runs, number of threads per block>(init, n, dev_id)
    reduce<int, 5, 128>(1, n, dev);
  }catch(std::runtime_error& e){
    std::cerr << e.what() << "\n";
    CHECK_CUDA( hipDeviceReset() ); // always call this at the end of your CUDA program
    return 1;
  }
  CHECK_CUDA( hipDeviceReset() ); // always call this at the end of your CUDA program
  return 0;
}
