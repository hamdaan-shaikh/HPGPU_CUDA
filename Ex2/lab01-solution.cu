
// nvcc lab01.cu -o kernel -std=c++11

#define CHECK_CUDA(cmd) {hipError_t error = cmd; if(error!=hipSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", hipGetErrorString(error));}}



#include <hip/hip_runtime.h>
#include <iostream>

int main(int, char**)
{
  const size_t length = 1000000;

  int* a_host = new int[length];
  int* b_host = new int[length];
  int *a_device = 0, *b_device = 0;
  int size = length*sizeof(int);

  for(int i=0; i<length; ++i)
    // initialize a_host
    a_host[i]=i;

  // allocate GPU memory on for a and b
  CHECK_CUDA ( hipMalloc(&a_device,size) ); 
  CHECK_CUDA ( hipMalloc(&b_device,size) );
  
  // data transfer a_host -> a_device
  CHECK_CUDA ( hipMemcpy(a_device, a_host, size, hipMemcpyHostToDevice) );
  
  // data copy (via data transfer) on device
  CHECK_CUDA ( hipMemcpy(b_device, a_device, size, hipMemcpyDeviceToDevice) );
  
  // data transfer b_device -> b_host
  CHECK_CUDA ( hipMemcpy(b_host, b_device, size, hipMemcpyDeviceToHost) );
  

  // free allocated GPU memory
  CHECK_CUDA ( hipFree(a_device) );
  CHECK_CUDA ( hipFree(b_device) );

  //a_host[42] = 0; // provoke an error

  
  for(int i=0; i<length; ++i)
    if(a_host[i] != b_host[i])
      std::cout << "Mismatch at: " << i << "\n";


  delete[] a_host;
  delete[] b_host;

  CHECK_CUDA( hipDeviceReset() ); // needed for clean exit in profilers/debuggers
  return 0;
}
